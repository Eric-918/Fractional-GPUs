#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

#ifndef USE_FGPU

/* With FGPU, can't use closed source GPU libraries (i.e. can't use cublas) */
template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

#if 0 /* Currently not in use in the code-base. No equivalent in FGPU currently */
template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X,
                           hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X,
                            hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}
#endif

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

#else  /* USE_FGPU */


#include <hip/hip_cooperative_groups.h>

/* TODO: Use shfl_down to reduce summation time */
/* Takes sum of all values of threads in a block and do one atomic addition */
template <typename Dtype>
__device__ void reduce_sum(cooperative_groups::thread_group g, Dtype val, Dtype *out)
{
    int lane = g.thread_rank();
    __shared__ Dtype temp [CAFFE_CUDA_NUM_THREADS];

    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2)
    {
        temp[lane] = val;
        g.sync(); // wait for all threads to store
        if(lane < i) 
            val += temp[lane + i];
        g.sync(); // wait for all threads to load
    }

    if (g.thread_rank() == 0)
        atomicAdd(out, val);
}

template <typename Dtype, size_t BLOCK_SIZE> 
__global__ void gemm_kernel(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int hA, const int wA, 
        const int hB, const int wB, const Dtype *A, const Dtype *B, 
        Dtype *C, const Dtype alpha, const Dtype beta)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aRow = BLOCK_SIZE * by;
    int aColBegin = 0;

    // Index of the last sub-matrix of A processed by the block
    int aColEnd = wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aColStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bRowBegin = 0;
    int bCol = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bRowStep  = BLOCK_SIZE;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B    
    // required to compute the block sub-matrix
    for (int aCol = aColBegin, bRow = bRowBegin, istep=0;
         aCol <= aColEnd;
         aCol += aColStep, bRow += bRowStep, ++istep)
    {
        int aIndex, bIndex;

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        aIndex = (TransA == CblasNoTrans) ? (wA * (aRow + ty) + aCol + tx) :
            (hA * (aCol + tx) + aRow + ty);

        bIndex = (TransB == CblasNoTrans) ? (wB * (bRow + ty) + bCol + tx) :
            (hB * (bCol + tx) + bRow + ty);

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        if ((aRow + ty < hA) && (aCol + tx < wA))
            As[ty][tx] = A[aIndex];
        else
            As[ty][tx] = 0;

        if ((bRow + ty < hB) && (bCol + tx < wB))
            Bs[ty][tx] = B[bIndex];
        else
            Bs[ty][tx] = 0;

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int cRow = BLOCK_SIZE * by + ty;
    int cCol = BLOCK_SIZE * bx + tx;
    int cIndex = wB * cRow + cCol;

    if ((cRow < hA) && (cCol < wB))
        C[cIndex] = alpha * Csub + beta * C[cIndex];
}

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float *A, const float *B, const float beta,
    float *C) {

  const int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grids((N + threads.x - 1) / threads.x, (M + threads.y - 1) / threads.y);
  gemm_kernel<float, block_size><<<grids, threads>>>(TransA, TransB, 
          M, K, K, N, A, B, C, alpha, beta);
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double *A, const double *B, const double beta,
    double *C) {

  const int block_size = 32;
  dim3 threads(block_size, block_size);
  dim3 grids((N + threads.x - 1) / threads.x, (M + threads.y - 1) / threads.y);
  gemm_kernel<double, block_size><<<grids, threads>>>(TransA, TransB, 
          M, K, K, N, A, B, C, alpha, beta);
}

template <typename Dtype> 
__global__ void gemv_kernel(const CBLAS_TRANSPOSE TransA,
    const int hA, const int wA, const Dtype *A, const Dtype *x, 
        Dtype *y, const Dtype alpha, const Dtype beta)
{
    int row = blockIdx.x;
    Dtype sum = 0;
    for (int col = threadIdx.x; col < wA; col += blockDim.x) {
        if (TransA == CblasNoTrans)
            sum += A[wA * row + col] * x[col];
        else
            sum += A[hA * col + row] * x[col];
    }

    sum = alpha * sum;

    cooperative_groups::thread_group  g = cooperative_groups::this_thread_block();
    
    if (g.thread_rank() == 0)
        y[row] += (beta - 1) * y[row];

    g.sync();

    reduce_sum<Dtype>(g, sum, &y[row]);
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float *A, const float *x,
    const float beta, float *y) {
    if (TransA == CblasNoTrans) {
        gemv_kernel<float><<<M, CAFFE_CUDA_NUM_THREADS>>>(
          TransA, M, N, A, x, y, alpha, beta);
    } else {
        gemv_kernel<float><<<N, CAFFE_CUDA_NUM_THREADS>>>(
          TransA, N, M, A, x, y, alpha, beta);
    }
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double *A, const double *x,
    const double beta, double *y) {
    if (TransA == CblasNoTrans) {
        gemv_kernel<double><<<M, CAFFE_CUDA_NUM_THREADS>>>(
          TransA, M, N, A, x, y, alpha, beta);
    } else {
        gemv_kernel<double><<<N, CAFFE_CUDA_NUM_THREADS>>>(
          TransA, N, M, A, x, y, alpha, beta);
    }
}

template <typename Dtype>
__global__ void axpy_kernel(const int n, const Dtype alpha, const Dtype *x, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha * x[index];
  }
}

template <>
void caffe_gpu_axpy(int n, float alpha, float const *x, float *y)
{
    axpy_kernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, alpha, x, y);
}

template <>
void caffe_gpu_axpy<double>(int n, double alpha, double const *x, double *y)
{
    axpy_kernel<double><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, alpha, x, y);
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <typename Dtype>
__global__ void scal_kernel(const int n, const Dtype alpha, Dtype *x) {
  CUDA_KERNEL_LOOP(index, n) {
    x[index] *= alpha;
  }
}

template <>
void caffe_gpu_scal<float>(const int n, float alpha, float *x)
{
    scal_kernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, alpha, x);
}


template <>
void caffe_gpu_scal<double>(const int n, double alpha, double *x)
{
    scal_kernel<double><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, alpha, x);
}

static void *initialize_temp_variable(void)
{
    void *temp;

    CUDA_CHECK(hipMalloc(&temp, sizeof(double)));
    CUDA_CHECK(hipMemset(temp, 0, sizeof(double)));

    return temp;
}


static void deinitialize_temp_variable(void *temp)
{
    CUDA_CHECK(hipFree(temp));
}


template <typename Dtype>
__global__ void dot_kernel(const int n, const Dtype *x, const Dtype *y, Dtype *out) {
  Dtype sum = 0;
  CUDA_KERNEL_LOOP(index, n) {
    sum += x[index] * y[index];
  }
  
  cooperative_groups::thread_group g = cooperative_groups::this_thread_block();
  reduce_sum<Dtype>(g, sum, out);
}

template <>
void caffe_gpu_dot<float>(const int n, const float *x, const float *y, float *out)
{
    void *tempResult = initialize_temp_variable();
    dot_kernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, (float *)tempResult);
    CUDA_CHECK(hipMemcpy(out, tempResult, sizeof(float), hipMemcpyDefault));
    deinitialize_temp_variable(tempResult);
}

template <>
void caffe_gpu_dot<double>(const int n, const double *x, const double *y, double *out)
{
    void *tempResult = initialize_temp_variable();
    dot_kernel<double><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, (double *)tempResult);
    CUDA_CHECK(hipMemcpy(out, tempResult, sizeof(double), hipMemcpyDefault));
    deinitialize_temp_variable(tempResult);
}

template <typename Dtype>
__global__ void asum_kernel(const int n, const Dtype *x, Dtype *out) {
  Dtype sum = 0;
  CUDA_KERNEL_LOOP(index, n) {
    sum += fabs(x[index]);
  }
  
  cooperative_groups::thread_group g = cooperative_groups::this_thread_block();
  reduce_sum<Dtype>(g, sum, out);
}

template <>
void caffe_gpu_asum<float>(const int n, const float *x, float *out)
{
    void *tempResult = initialize_temp_variable();
    asum_kernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, (float *)tempResult);
    CUDA_CHECK(hipMemcpy(out, tempResult, sizeof(float), hipMemcpyDefault));
    deinitialize_temp_variable(tempResult);
}

template <>
void caffe_gpu_asum<double>(const int n, const double *x, double *out)
{
    void *tempResult = initialize_temp_variable();
    asum_kernel<double><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, (double *)tempResult);
    CUDA_CHECK(hipMemcpy(out, tempResult, sizeof(double), hipMemcpyDefault));
    deinitialize_temp_variable(tempResult);
}

template <typename Dtype>
__global__ void scale_kernel(const int n, const Dtype alpha, const Dtype *x, 
        Dtype *y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha * x[index];
  }
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
    scale_kernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
            n, alpha, x, y);
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
    scale_kernel<double><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
            n, alpha, x, y);
}
#endif /* USE_FGPU */

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}


template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sqrt_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = sqrt(a[index]);
  }
}

template <>
void caffe_gpu_sqrt<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_sqrt<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void sign_kernel(const int n, const Dtype* x, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = (Dtype(0) < x[index]) - (x[index] < Dtype(0));
  }
}

template <>
void caffe_gpu_sign<float>(const int N, const float* x, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, x, y);
}

template <>
void caffe_gpu_sign<double>(const int N, const double* x, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, x, y);
}

template <typename Dtype>
__global__ void sgnbit_kernel(const int n, const Dtype* x, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = signbit(x[index]);
  }
}

template <>
void caffe_gpu_sgnbit<float>(const int N, const float* x, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sgnbit_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, x, y);
}

template <>
void caffe_gpu_sgnbit<double>(const int N, const double* x, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sgnbit_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, x, y);
}

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe
