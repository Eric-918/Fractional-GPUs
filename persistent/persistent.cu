#include "hip/hip_runtime.h"
/* This file contains API for persistent kernels */
#include <assert.h>
#include <dlfcn.h>
#include <fcntl.h>
#include <inttypes.h>
#include <limits.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#include <map>

#include <hip/hip_runtime.h>

#include <common.h>
#include <fractional_gpu.h>
#include <persistent.h>

/* Name of the shared files */
#define FGPU_SHMEM_NAME             "fgpu_shmem"
#define FGPU_SHMEM_HOST_NAME        "fgpu_host_shmem"

/* File used by MPS */
#define FGPU_MPS_CONTROL_NAME       "/tmp/nvidia-mps/control"

/* TODO: Add support for multiple devices */
/* TODO: Add proper logging mechanism */

/* Currently only the very first device is used */
#define FGPU_DEVICE_NUMBER  0

/* Look into hipHostRegister and hipHostGetFlags and hipInit*/

typedef struct kernel_info {
    int tag;
    int color;
    int num_pblocks_launched;
    bool is_done;
    int return_code;
} kernel_info_t;

static volatile fgpu_indicators_t *h_indicators;

/* Can't share streams between process - As limitation by CUDA library/driver */
hipStream_t streams[FGPU_MAX_NUM_COLORS];

/* This structure contains all host side information for persistent thread ctx */
typedef struct fgpu_host_ctx {
  
    int device;
    int num_colors;
    int num_sm;
    int max_num_threads_per_sm;

    std::pair<uint32_t, uint32_t> color_to_sms[FGPU_MAX_NUM_COLORS];

    volatile fgpu_indicators_t *d_indicators;

    fgpu_bindexes_t *d_bindexes;
    
    bool cur_indexes[FGPU_MAX_NUM_COLORS];
    int last_color;
    int last_tags[FGPU_MAX_NUM_COLORS];
    int cur_tag;

    /* Kernel tag -> Info kernels */
    /* TODO: Make this a map in shared memory - See boost interprocess lib */
    kernel_info_t tag_to_info[FGPU_MAX_PENDING_TASKS];
//    static std::map<int, kernel_info_t> tag_to_info;

} fgpu_host_ctx_t;

/* Host side context */
static fgpu_host_ctx_t *g_host_ctx;

/* Shared memories file descriptor */
static int shmem_fd = -1;
static int shmem_host_fd = -1;

/* Checks if MPS is enabled */
static bool is_mps_enabled(void)
{
    int ret;
    struct stat st;

    ret = stat(FGPU_MPS_CONTROL_NAME, &st);
    if (ret < 0)
        return false;

    return true;
}

/* Sets color info per device */
static int init_color_info(fgpu_host_ctx_t *host_ctx,
        const hipDeviceProp_t *device_prop)
{
    int num_colors;
    int num_sm = device_prop->multiProcessorCount;
    int sm_per_color;

    /* 
     * Colors depends on the memory hieracy on device and limitations of
     * coloing. Presently only 2 colors are supported in case of userspace
     * coloring.
     */
    if (strcmp(device_prop->name, "GeForce GTX 1070") == 0) {
        
        num_colors = 2;
        host_ctx->num_colors = num_colors;

    } else {
        /* All CUDA devices are not currently supported */
        fprintf(stderr, "Unknown CUDA device\n");
        return -1;
    }

    assert(FGPU_MAX_NUM_COLORS >= num_colors);

    /*
     * Due to integer division, all colors might not be balanced perfectly
     * Currently we are treating all colors equally. This is not neccesary
     */
    sm_per_color = num_sm / num_colors;

    if (sm_per_color == 0) {
        fprintf(stderr, "Too few SMs/Too many colors\n");
        return -1;
    }

    printf("Device: \"%s\", Number of Colors:%d\n", device_prop->name, num_colors);
    for (int i = 0; i < num_colors; i++) {
        int start_sm;
        int end_sm;

        start_sm = i * sm_per_color;
        end_sm = (i + 1) * sm_per_color - 1;
        if (i == num_colors - 1)
            end_sm = num_sm - 1;

        host_ctx->color_to_sms[i] = std::make_pair(start_sm, end_sm);
        if (i == 0)
            host_ctx->color_to_sms[i] = std::make_pair(0, 14);
        else
            host_ctx->color_to_sms[i] = std::make_pair(15, 15);
        printf("Color:%d, SMs:(%d->%d)\n", i, start_sm, end_sm);
    }

    return 0;
}

/* Sets the device to first available device */
static int init_device_info(fgpu_host_ctx_t *host_ctx)
{
    int deviceCount = 0;
    hipDeviceProp_t device_prop;
    size_t max_threads;

    int ret = gpuErrCheck(hipGetDeviceCount(&deviceCount));
    if (ret < 0)
        return ret;

    if (deviceCount == 0) {
        fprintf(stderr, "Couldn't find any CUDA devices\n");    
        return -1;
    }

    assert(deviceCount > FGPU_DEVICE_NUMBER);

    ret = gpuErrCheck(hipSetDevice(FGPU_DEVICE_NUMBER));
    if (ret < 0)
        return ret;

    ret = gpuErrCheck(hipGetDeviceProperties(&device_prop, FGPU_DEVICE_NUMBER));
    if (ret < 0)
        return ret;

    max_threads = device_prop.maxThreadsPerMultiProcessor *
        device_prop.multiProcessorCount;

    if (max_threads > FGPU_MAX_NUM_PBLOCKS * FGPU_MIN_BLOCKDIMS) {
        fprintf(stderr, "Too many SMs/Threads in CUDA device\n");
        return -1;
    }

    if (device_prop.warpSize != FGPU_MIN_BLOCKDIMS) {
        fprintf(stderr, "Warp size of CUDA device is not correct\n");
        return -1;
    }

    host_ctx->device = FGPU_DEVICE_NUMBER;
    host_ctx->num_sm = device_prop.multiProcessorCount;
    host_ctx->max_num_threads_per_sm = device_prop.maxThreadsPerMultiProcessor;

    ret = init_color_info(host_ctx, &device_prop);
    if (ret < 0)
        return ret;
    
    return 0;
}

/* Initialize (by server) */
int fgpu_server_init(void)
{
    int ret = 0;
    size_t shmem_size;
    size_t page_size;
    hipCtx_t driver_ctx;

    ret = gpuDriverErrCheck(hipInit(0));
    if (ret < 0)
        goto err;
    
    if (!is_mps_enabled()) {
        fprintf(stderr, "MPS is not enabled\n");
        goto err;
    }

    /* Create the shared memory */
    ret = shmem_fd = shm_open(FGPU_SHMEM_NAME,
            O_RDWR | O_CREAT | O_EXCL, S_IRUSR | S_IWUSR);
    if (ret < 0) {
        fprintf(stderr, "Couldn't open shmem\n");
        goto err;
    }

    page_size = sysconf(_SC_PAGE_SIZE);

    shmem_size = ROUND_UP(sizeof(fgpu_host_ctx_t), page_size);

    ret = ftruncate(shmem_fd, shmem_size);
    if (ret < 0) {
        fprintf(stderr, "Can't truncate shmem file\n");
        ret = -1;
        goto err;
    }

    g_host_ctx = (fgpu_host_ctx_t *)mmap(NULL, shmem_size,
                    PROT_READ | PROT_WRITE, MAP_SHARED, shmem_fd, 0);
    if (g_host_ctx == NULL) {
        fprintf(stderr, "Can't map shmem\n");
        ret = -1;
        goto err;
    }

    /* Allocate bindexes on device memory */
    ret = gpuErrCheck(hipMalloc(&g_host_ctx->d_bindexes,
                sizeof(fgpu_bindexes_t)));
    if (ret < 0)
        goto err;

    assert(FGPU_MAX_NUM_COLORS > 0);

    ret = gpuErrCheck(hipMemset(g_host_ctx->d_bindexes, 0, sizeof(fgpu_bindexes_t)));
    if (ret < 0)
        goto err;


    ret = shmem_host_fd = shm_open(FGPU_SHMEM_HOST_NAME,
            O_RDWR | O_CREAT | O_EXCL, S_IRUSR | S_IWUSR);
    if (ret < 0) {
        fprintf(stderr, "Couldn't open shmem\n");
        goto err;
    }

    shmem_size = ROUND_UP(sizeof(fgpu_indicators_t), page_size);

    ret = ftruncate(shmem_host_fd, shmem_size);
    if (ret < 0) {
        fprintf(stderr, "Can't truncate shmem (host) file\n");
        ret = -1;
        goto err;
    }

    h_indicators = (volatile fgpu_indicators_t *)mmap(NULL, shmem_size,
                    PROT_READ | PROT_WRITE, MAP_SHARED, shmem_host_fd, 0);
    if (h_indicators == NULL) {
        fprintf(stderr, "Can't map shmem\n");
        ret = -1;
        goto err;
    }

    ret = gpuDriverErrCheck(hipHostRegister((void *)h_indicators, shmem_size,
                hipHostRegisterPortable | hipHostRegisterMapped));
    if (ret < 0)
        goto err;
    
    ret = gpuErrCheck(hipHostGetDevicePointer(&g_host_ctx->d_indicators,
                (void *)h_indicators, 0));
    if (ret < 0)
        goto err;

    memset((void *)h_indicators, 0, sizeof(fgpu_indicators_t));

    g_host_ctx->last_color = -1;

    for (int i = 0; i < FGPU_MAX_NUM_COLORS; i++)
        g_host_ctx->last_tags[i] = -1;

    for (int i = 0; i < FGPU_MAX_PENDING_TASKS; i++)
        g_host_ctx->tag_to_info[i].tag = -1;

    ret = init_device_info(g_host_ctx);
    if (ret < 0)
        goto err;

    /* 
     * Server doesn't need to create streams because server is not launching
     * processes
     */
    return 0;

err:
    fgpu_server_deinit();
    return ret;
}

/* Deinitializes */
void fgpu_server_deinit(void)
{
    printf("Server Terminating. Waiting for device to be free\n");
    gpuErrCheck(hipDeviceSynchronize());

    if (g_host_ctx) {
        if (g_host_ctx->d_bindexes != NULL)
            hipFree((void *)g_host_ctx->d_bindexes);
    }
    
    if (h_indicators != NULL) {
        hipHostUnregister((void *)h_indicators);
        hipHostFree((void *)h_indicators);
    }

    if (shmem_host_fd > 0)
        close(shmem_host_fd);
    
    if (shmem_fd > 0)
        close(shmem_fd);

    /* Remove links so that can be reused */
    shm_unlink(FGPU_SHMEM_HOST_NAME);
    shm_unlink(FGPU_SHMEM_NAME);
}

/* Initialization for non-server */
int fgpu_init(void)
{
    int ret;
    size_t page_size;
    size_t shmem_size;

    if (!is_mps_enabled()) {
        fprintf(stderr, "MPS is not enabled\n");
        goto err;
    }


    /* Create the shared memory */
    ret = shmem_fd = shm_open(FGPU_SHMEM_NAME, O_RDWR, S_IRUSR | S_IWUSR);
    if (ret < 0) {
        fprintf(stderr, "Couldn't open shmem\n");
        goto err;
    }

    page_size = sysconf(_SC_PAGE_SIZE);

    shmem_size = ROUND_UP(sizeof(fgpu_host_ctx_t), page_size);
    g_host_ctx = (fgpu_host_ctx_t *)mmap(NULL, shmem_size,
                    PROT_READ | PROT_WRITE, MAP_SHARED, shmem_fd, 0);
    if (g_host_ctx == NULL) {
        fprintf(stderr, "Can't map shmem\n");
        ret = -1;
        goto err;
    }

    ret = shmem_host_fd = shm_open(FGPU_SHMEM_HOST_NAME, O_RDWR, S_IRUSR | S_IWUSR);
    if (ret < 0) {
        fprintf(stderr, "Couldn't open shmem\n");
        goto err;
    }

    shmem_size = ROUND_UP(sizeof(fgpu_indicators_t), page_size);

    h_indicators = (volatile fgpu_indicators_t *)mmap(NULL, shmem_size,
            PROT_READ | PROT_WRITE, MAP_SHARED, shmem_host_fd, 0);
    if (h_indicators == NULL) {
        fprintf(stderr, "Can't map shmem (host pinned)\n");
        ret = -1;
        goto err;
    }

    /* Create seperate streams for each color */
    for (int i = 0; i < g_host_ctx->num_colors; i++) {
        ret = gpuErrCheck(hipStreamCreate(&streams[i]));
        if (ret < 0)
            goto err;
    }

    return 0;

err:
    fgpu_server_deinit();
    return ret;
}

void fgpu_deinit(void)
{
    for (int i = 0; i < g_host_ctx->num_colors; i++) {
        hipStreamDestroy(streams[i]);
    }

    if (shmem_host_fd > 0)
        close(shmem_host_fd);
    
    if (shmem_fd > 0)
        close(shmem_fd);
}


/* Wait for last launched kernel to be completely started */
static void wait_for_last_start(void)
{
    if (g_host_ctx->last_color >= 0) {
        int last_tag = g_host_ctx->last_tags[g_host_ctx->last_color];

        if (last_tag >= 0) {

            kernel_info_t last_info;

            last_info = g_host_ctx->tag_to_info[last_tag];
            
            /* 
             * Need to wait for the last launched kernel to indicate all blocks
             * have been launched.
             */
            assert(last_info.is_done == false);
            for (int i = 0; i < last_info.num_pblocks_launched; i++) {
                while (!h_indicators->indicators[i].started[last_info.color]);
                h_indicators->indicators[i].started[last_info.color] = false;
            }
            g_host_ctx->last_color = -1;
        }
    }
}

/* Wait for last launched kernel of a specific color to be completed */
static int wait_for_last_complete(int color)
{
    int ret = 0;;

//    gpuErrCheck(hipStreamSynchronize(streams[color]));
//    last_color = -1;
//    return 0;

    if (g_host_ctx->last_tags[color] >= 0) {
        kernel_info_t *last_info;
        ret = gpuErrCheck(hipStreamSynchronize(streams[color]));

        last_info = &g_host_ctx->tag_to_info[g_host_ctx->last_tags[color]];
        last_info->is_done = true;
        last_info->return_code = ret;
        g_host_ctx->last_tags[color] = -1;

        if (g_host_ctx->last_color == color)
            g_host_ctx->last_color = -1;
    }

    return ret;

}
/* Prepare ctx before launch */
int fgpu_prepare_launch_kernel(fgpu_dev_ctx_t *ctx, uint3 *_gridDim, hipStream_t **stream)
{
    uint32_t num_blocks;
    uint32_t num_threads;
    uint32_t num_pblocks;
    int color = ctx->color;
    int ret;
    kernel_info_t info;

    int tag;
    if (color >= g_host_ctx->num_colors || color < 0)
        return -1;

    num_blocks = ctx->gridDim.x * ctx->gridDim.y * ctx->gridDim.z;
    if (num_blocks == 0)
        return -1;

    num_threads = ctx->blockDim.x * ctx->blockDim.y * ctx->blockDim.z;
    if (num_threads == 0 || num_threads > g_host_ctx->max_num_threads_per_sm)
        return -1;

    /* Num threads should be power of 2 */
    /* TODO: Relax this constraint */
    if (num_threads & (num_threads - 1) != 0)
        return -1;

    if (num_threads < FGPU_MIN_BLOCKDIMS)
        return -1;
    
    ret = wait_for_last_complete(color);
    if (ret < 0)
        return ret;

    wait_for_last_start();

    num_pblocks =
        (g_host_ctx->num_sm * g_host_ctx->max_num_threads_per_sm) / num_threads;

    ctx->num_blocks = num_blocks;
    ctx->index = g_host_ctx->cur_indexes[color];
    g_host_ctx->cur_indexes[color] ^= 1;   /* Toggle the index */
    ctx->d_indicators = g_host_ctx->d_indicators;
    ctx->d_bindex = g_host_ctx->d_bindexes;
    ctx->start_sm = g_host_ctx->color_to_sms[color].first;
    ctx->end_sm = g_host_ctx->color_to_sms[color].second;

    tag = ++g_host_ctx->cur_tag % FGPU_MAX_PENDING_TASKS;
    info.color = color;
    info.num_pblocks_launched = num_pblocks;
    info.is_done = false;
    g_host_ctx->tag_to_info[tag] = info;

    g_host_ctx->last_tags[color] = tag;
    g_host_ctx->last_color = color;

    _gridDim->x = num_pblocks;
    _gridDim->y = 1;
    _gridDim->z = 1;
    *stream = &streams[color];

    return tag;
}

int fgpu_wait_for_kernel(int tag)
{
//    std::map<int, kernel_info_t>::iterator it = g_host_ctx->tag_to_info.find(tag);
    kernel_info_t info;
    int ret;

//    if (it == g_host_ctx->tag_to_info.end())
//        return -1;

//    info = it->second;
    info = g_host_ctx->tag_to_info[tag];

    if (info.tag == -1)
        return -1;

    if (info.is_done) {
        g_host_ctx->tag_to_info[tag].tag = -1;
//        g_host_ctx->tag_to_info.erase(tag);
        return info.return_code;
    }

    assert(tag == g_host_ctx->last_tags[info.color]);

    ret = wait_for_last_complete(info.color);

    g_host_ctx->tag_to_info[tag].tag = -1;
//    g_host_ctx->tag_to_info.erase(tag);

    return ret;
}

