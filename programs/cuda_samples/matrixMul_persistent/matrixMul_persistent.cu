#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>

#include <fractional_gpu.hpp>
#include <fractional_gpu_cuda.cuh>

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> 
FGPU_DEFINE_KERNEL(matrixMulCUDA, float *C, float *A, float *B, int wA, int wB)
{
    fgpu_dev_ctx_t *ctx;
    uint3 _blockIdx;
    ctx = FGPU_DEVICE_INIT();

    FGPU_FOR_EACH_DEVICE_BLOCK(_blockIdx) {
        // Block index
        int bx = _blockIdx.x;
        int by = _blockIdx.y;

        // Thread index
        int tx = threadIdx.x;
        int ty = threadIdx.y;

        // Index of the first sub-matrix of A processed by the block
        int aBegin = wA * BLOCK_SIZE * by;

        // Index of the last sub-matrix of A processed by the block
        int aEnd   = aBegin + wA - 1;

       // Step size used to iterate through the sub-matrices of A
        int aStep  = BLOCK_SIZE;

        // Index of the first sub-matrix of B processed by the block
        int bBegin = BLOCK_SIZE * bx;

        // Step size used to iterate through the sub-matrices of B
        int bStep  = BLOCK_SIZE * wB;

        // Csub is used to store the element of the block sub-matrix
        // that is computed by the thread
        float Csub = 0;

        // Loop over all the sub-matrices of A and B
        // required to compute the block sub-matrix
        for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep)
        {

            // Declaration of the shared memory array As used to
            // store the sub-matrix of A
            __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

            // Declaration of the shared memory array Bs used to
            // store the sub-matrix of B
            __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

            // Load the matrices from device memory
            // to shared memory; each thread loads
            // one element of each matrix
            As[ty][tx] = FGPU_COLOR_LOAD(ctx, &A[a + wA * ty + tx]);
            Bs[ty][tx] = FGPU_COLOR_LOAD(ctx, &B[b + wB * ty + tx]);

            // Synchronize to make sure the matrices are loaded
            __syncthreads();

            // Multiply the two matrices together;
            // each thread computes one element
            // of the block sub-matrix
#pragma unroll

            for (int k = 0; k < BLOCK_SIZE; ++k)
            {
                Csub += As[ty][k] * Bs[k][tx];
            }

            // Synchronize to make sure that the preceding
            // computation is done before loading two new
            // sub-matrices of A and B in the next iteration
            __syncthreads();
        }

        // Write the block sub-matrix to device memory;
        // each thread writes one element
        int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
        FGPU_COLOR_STORE(ctx, &C[c + wB * ty + tx], Csub);
    } FGPU_FOR_EACH_END;
}



void constantInit(float *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}


/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiply(void)
{
    int block_size = 32;
    dim3 dimsA(10 * block_size, 10 * block_size, 1);
    dim3 dimsB(20 * block_size, 10 * block_size, 1);

    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;

    // Initialize host memory
    const float valB = 0.01f;

    // Allocate device memory
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);

    int ret;
    
    ret = fgpu_memory_allocate((void **) &h_A, mem_size_A);

    if (ret < 0)
    {
        printf("fgpu_memory_allocate h_A returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_memory_allocate((void **) &h_B, mem_size_B);

    if (ret < 0)
    {
        printf("fgpu_memory_allocate h_B returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_memory_allocate((void **) &h_C, mem_size_C);

    if (ret < 0)
    {
        printf("fgpu_memory_allocate h_C returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // copy host memory to device
    ret = fgpu_memory_prefetch_to_device_async(h_A, mem_size_A);

    if (ret < 0)
    {
        printf("fgpu_memory_prefetch_to_device_async (h_A) returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_memory_prefetch_to_device_async(h_B, mem_size_B);

    if (ret < 0)
    {
        printf("fgpu_memory_prefetch_to_device_async (h_B) returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_memory_prefetch_to_device_async(h_C, mem_size_C);

    if (ret < 0)
    {
        printf("fgpu_memory_prefetch_to_device_async (h_C) returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_color_stream_synchronize();
    if (ret < 0)
    {
        printf("fgpu_color_stream_synchronize returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    /* Get the device pointers */
    ret = fgpu_memory_get_device_pointer((void **)&d_A, h_A);
    if (ret < 0) {
        printf("fgpu_memory_get_device_pointer (d_A) returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_memory_get_device_pointer((void **)&d_B, h_B);
    if (ret < 0) {
        printf("fgpu_memory_get_device_pointer (d_B) returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_memory_get_device_pointer((void **)&d_C, h_C);
    if (ret < 0) {
        printf("fgpu_memory_get_device_pointer (d_C) returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Execute the kernel
    int nIter = 10000;

    double start, total;
    pstats_t stats;

    // Init
    for (int j = 0; j < nIter; j++)
    {
        start = dtime_usec(0);

        if (block_size == 16)
        {
            ret = FGPU_LAUNCH_KERNEL(grid, threads, 0, matrixMulCUDA<16>, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            ret = FGPU_LAUNCH_KERNEL(grid, threads, 0, matrixMulCUDA<32>, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        if (ret < 0)
            return ret;
	
	ret = fgpu_color_stream_synchronize();
    	if (ret < 0)
        	return ret;

        total = dtime_usec(start);
        printf("Time:%f, BlockSize:%d, dimA.x:%d, dimA.y:%d, dimB.x:%d, dimB.y:%d\n", total, block_size, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
    }

    pstats_init(&stats);
    start = dtime_usec(0);
    for (int j = 0; j < nIter; j++)
    {
        double sub_start = dtime_usec(0);
        if (block_size == 16)
        {
            ret = FGPU_LAUNCH_KERNEL(grid, threads, 0, matrixMulCUDA<16>, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            ret = FGPU_LAUNCH_KERNEL(grid, threads, 0, matrixMulCUDA<32>, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        if (ret < 0)
            return ret;
        pstats_add_observation(&stats, dtime_usec(sub_start));
    }

    ret = fgpu_color_stream_synchronize();
    if (ret < 0)
        return ret;

    total = dtime_usec(start);
    pstats_print(&stats);

    // Compute and print the performance
    double msecPerMatrixMul = total / nIter / 1000;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.6f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Terminate - To overlap wth application running in other color for benchmarking
    for (int j = 0; j < nIter; j++)
    {

        if (block_size == 16)
        {
            ret = FGPU_LAUNCH_KERNEL(grid, threads, 0, matrixMulCUDA<16>, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            ret = FGPU_LAUNCH_KERNEL(grid, threads, 0, matrixMulCUDA<32>, d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        if (ret < 0)
            return ret;

	    ret = fgpu_color_stream_synchronize();
    	if (ret < 0)
        	return ret;

    }

    // Copy result from device to host
    ret = fgpu_memory_prefetch_from_device_async(h_C, mem_size_C);
    if (ret < 0)
    {
        printf("fgpu_memory_prefetch_from_device_async (h_C) returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE);
    }

    ret = fgpu_color_stream_synchronize();
    if (ret < 0)
    {
        printf("fgpu_color_stream_synchronize returned error %s (code %d), line(%d)\n", strerror(errno), ret, __LINE__);
        exit(EXIT_FAILURE        );
    }

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6 ; // machine zero

    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err/abs_val/dot_length ;

        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    fgpu_memory_free(h_A);
    fgpu_memory_free(h_B);
    fgpu_memory_free(h_C);

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}

int main(int argc, char *argv[])
{
    int ret;
    int color;

    if (argc != 2) {
        fprintf(stderr, "Insufficient number of arguments\n");
        exit(-1);
    }

    color = atoi(argv[1]);

    printf("Color selected:%d\n", color);

    ret = fgpu_init();
    if (ret < 0)
        return ret;

    ret = fgpu_set_color_prop(color, 128 * 1024 * 1024);
    if (ret < 0)
        return ret;

    ret = matrixMultiply();
    if (ret < 0)
        return ret;

    fgpu_deinit();

    return 0;
}
